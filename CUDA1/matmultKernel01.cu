#include "hip/hip_runtime.h"
///
/// matmultKernel00.cu
/// For COMS E6998 Spring 2023
/// Instructor: Parajit Dube and Kaoutar El Maghraoui
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman
/// Created: 2011-01-27
/// Last Modified: 2011-02-23 DVN
///
/// Multiplies two matrices using CUDA: A x B = C
///
/// Copy this file and modify the MatMultKernel device function for
/// each of your experiments. 
///

#include "matmultKernel.h"
#include <stdio.h>

// Define a gpu kernel to perform matrix multiplication
// of A x B = C.
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){ 
  // matrix blocks
  float *Asub, *Bsub, *Csub;
  // Putting these into registers speeds access.
  int thread_row = threadIdx.y;
  int thread_col = threadIdx.x;
  int block_row = blockIdx.y;
  int block_col = blockIdx.x;

  // Each THREAD BLOCK computes one sub matrix Csub of C
  // EACH THREAD creates its own matrix descriptor Csub
  Csub = &C.elements[C.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * block_col];

  // Each thread computes four elements of Csub in its copy of CValue
  float Cvalue0 = 0;
  float Cvalue1 = 0;
  float Cvalue2 = 0;
  float Cvalue3 = 0;

  // Loop over all sub matrices in block_row of A and block_col of B
  // required to compute Csub. Block multiply each pair of sub matrices
  // and accumulate results
  for (int m = 0;  m < (A.width / FOOTPRINT_SIZE); ++m){
    // Get Asub and Bsub descriptors
    Asub = &A.elements[A.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * m];
    Bsub = &B.elements[B.stride * FOOTPRINT_SIZE * m + FOOTPRINT_SIZE * block_col];

    // Copy ELEMENTS OF  ASub and Bsub into shared memory
    // EACH THREAD loads ONE ELEMENT of ASub and ONE of Bsub
    // Notice: it does not need to be the element it requires to
    //         compute its Cvalue, as long as all elements are 
    //         collaboratively read. 

    // Notice: every thread declares shared_A and shared_B in shared memory
    //         even though a thread block has only one shared_A and one shared_B
    __shared__ float shared_A[FOOTPRINT_SIZE][FOOTPRINT_SIZE];
    __shared__ float shared_B[FOOTPRINT_SIZE][FOOTPRINT_SIZE];

    // Each thread copies 4 elements of shared_A and 4 elements of shared_B
    shared_A[thread_row][thread_col] = Asub[thread_row * A.stride + thread_col];
    shared_A[thread_row + FOOTPRINT_SIZE/2][thread_col] = Asub[(thread_row + FOOTPRINT_SIZE/2) * A.stride + thread_col];
    shared_A[thread_row][thread_col + FOOTPRINT_SIZE/2] = Asub[thread_row * A.stride + thread_col + FOOTPRINT_SIZE/2];
    shared_A[thread_row + FOOTPRINT_SIZE/2][thread_col + FOOTPRINT_SIZE/2] = Asub[(thread_row + FOOTPRINT_SIZE/2) * A.stride + thread_col + FOOTPRINT_SIZE/2];

    shared_B[thread_row][thread_col] = Bsub[thread_row * B.stride + thread_col];
    shared_B[thread_row + FOOTPRINT_SIZE/2][thread_col] = Bsub[(thread_row + FOOTPRINT_SIZE/2) * B.stride + thread_col];
    shared_B[thread_row][thread_col + FOOTPRINT_SIZE/2] = Bsub[thread_row * B.stride + thread_col + FOOTPRINT_SIZE/2];
    shared_B[thread_row + FOOTPRINT_SIZE/2][thread_col + FOOTPRINT_SIZE/2] = Bsub[(thread_row + FOOTPRINT_SIZE/2) * B.stride + thread_col + FOOTPRINT_SIZE/2];

    // Synchronize to ensure all elements are read
    __syncthreads();

    // Do an inproduct of one row of shared_A and one col of shared_B
    // computing one Cvalue by accumulation
#pragma unroll
    for(int e=0; e<FOOTPRINT_SIZE; e++){
      Cvalue0 += shared_A[thread_row][e] * shared_B[e][thread_col];
      Cvalue1 += shared_A[thread_row][e] * shared_B[e][thread_col + FOOTPRINT_SIZE/2];
      Cvalue2 += shared_A[thread_row + FOOTPRINT_SIZE/2][e] * shared_B[e][thread_col];
      Cvalue3 += shared_A[thread_row + FOOTPRINT_SIZE/2][e] * shared_B[e][thread_col + FOOTPRINT_SIZE/2];
    }
       

    // Synchronize to ensure all Cvalues have been incremented
    // before reading in the next shared_A AND shared_B BLOCKS
    __syncthreads();
  }

  // Write Csub to GLOBAL memory.
  // Each thread writes its own cell value.
  Csub[thread_row * C.stride + thread_col] = Cvalue0;
  Csub[thread_row * C.stride + thread_col + FOOTPRINT_SIZE/2] = Cvalue1;
  Csub[(thread_row + FOOTPRINT_SIZE/2) * C.stride + thread_col] = Cvalue2;
  Csub[(thread_row + FOOTPRINT_SIZE/2) * C.stride + thread_col + FOOTPRINT_SIZE/2] = Cvalue3;
}