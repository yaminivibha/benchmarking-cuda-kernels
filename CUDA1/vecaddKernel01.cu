
#include <hip/hip_runtime.h>


__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        C[tid] = A[tid] + B[tid];
        tid += blockDim.x * gridDim.x;
    }
}