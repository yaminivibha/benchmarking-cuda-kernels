
#include <hip/hip_runtime.h>


__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}